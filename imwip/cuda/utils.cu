/*
:file:      utils.cu
:brief:     utils
:date:      1 AUG 2022
:author:    Jens Renders
            imec-Visionlab
            University of Antwerp
            jens.renders@uantwerpen.be
*/
#include <stdio.h>
#include <utils.hu>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


void setDevice(int device){
    gpuErrchk(hipSetDevice(device));
}