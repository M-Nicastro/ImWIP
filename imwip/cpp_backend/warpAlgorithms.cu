/*
:file:      warpAlgorithms.cu
:brief:     DVF based warping algorithms
:date:      20 DEC 2021
:author:    Jens Renders
            imec-Visionlab
            University of Antwerp
            jens.renders@uantwerpen.be
*/
#include <stdio.h>

#include <warpKernels.cu>
#include <warpAlgorithms.hu>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void warp2D(
        const float* f,
        const float* u,
        const float* v,
        float* fWarped,
        int degree,
        int shape0,
        int shape1
    ){

    size_t size = shape0 * shape1 * sizeof(float);

    // allocate vectors in device memory
    float *d_f, *d_u, *d_v, *d_fWarped;
    gpuErrchk(hipMalloc(&d_f, size));
    gpuErrchk(hipMalloc(&d_u, size));
    gpuErrchk(hipMalloc(&d_v, size));
    gpuErrchk(hipMalloc(&d_fWarped, size));

    // copy vectors from host memory to device memory
    gpuErrchk(hipMemcpy(d_f, f, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_u, u, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_v, v, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_fWarped, fWarped, size, hipMemcpyHostToDevice));

    // kernel invocation with 16*16 threads per block, and enough blocks
    // to cover the entire length of the vectors
    dim3 threadsPerBlock(16,16);
    dim3 numBlocks((shape0 + 15)/16, (shape1 + 15)/16);
    if(degree==1){
        linearWarp2DKernel<<<numBlocks, threadsPerBlock>>>(
            d_f,
            d_u,
            d_v,
            d_fWarped,
            shape0,
            shape1
        );
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }else if(degree==3){
        float coeffs[] = {
            #include "cubic_2D_coefficients.inc"
        };
        float *d_coeffs;
        gpuErrchk(hipMalloc(&d_coeffs, 16*16*sizeof(float)));
        gpuErrchk(hipMemcpy(d_coeffs, coeffs, 16*16*sizeof(float), hipMemcpyHostToDevice));
        cubicWarp2DKernel<<<numBlocks, threadsPerBlock>>>(
            d_f,
            d_u,
            d_v,
            d_fWarped,
            shape0,
            shape1,
            d_coeffs
        );
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        hipFree(d_coeffs);
    }else{
        throw "Only degree 1 and 3 are implemented";
    }

    // copy the result back to the host
    gpuErrchk(hipMemcpy(fWarped, d_fWarped, size, hipMemcpyDeviceToHost));

    // release the device memory
    hipFree(d_f);
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_fWarped);
}


void adjointWarp2D(
        const float* fWarped,
        const float* u,
        const float* v,
        float* f,
        int degree,
        int shape0,
        int shape1
    ){

    /*
    GPU implementation of 2D adjoint backward image warping along the DVF (u,v)
    with rectangular multivariate spline interpolation
    */


    size_t size = shape0 * shape1 * sizeof(float);

    // allocate vectors in device memory
    float *d_fWarped, *d_u, *d_v, *d_f;
    gpuErrchk(hipMalloc(&d_fWarped, size));
    gpuErrchk(hipMalloc(&d_u, size));
    gpuErrchk(hipMalloc(&d_v, size));
    gpuErrchk(hipMalloc(&d_f, size));

    // copy vectors from host memory to device memory
    gpuErrchk(hipMemcpy(d_fWarped, fWarped, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_u, u, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_v, v, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_f, f, size, hipMemcpyHostToDevice));

    // kernel invocation with 16*16 threads per block, and enough blocks
    // to cover the entire length of the vectors
    dim3 threadsPerBlock(16,16);
    dim3 numBlocks((shape0 + 15)/16, (shape1 + 15)/16);
    if(degree==1){
        adjointLinearWarp2DKernel<<<numBlocks, threadsPerBlock>>>(
            d_fWarped,
            d_u,
            d_v,
            d_f,
            shape0,
            shape1
        );
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }else if(degree==3){
        float coeffs[] = {
            #include "cubic_2D_coefficients.inc"
        };
        float *d_coeffs;
        gpuErrchk(hipMalloc(&d_coeffs, 16*16*sizeof(float)));
        gpuErrchk(hipMemcpy(d_coeffs, coeffs, 16*16*sizeof(float), hipMemcpyHostToDevice));
        adjointCubicWarp2DKernel<<<numBlocks, threadsPerBlock>>>(
            d_fWarped,
            d_u,
            d_v,
            d_f,
            shape0,
            shape1,
            d_coeffs
        );
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        hipFree(d_coeffs);
    }else{
        throw "Only degree 1 and 3 are implemented";
    }

    // copy the result back to the host
    gpuErrchk(hipMemcpy(f, d_f, size, hipMemcpyDeviceToHost));

    // release the device memory
    hipFree(d_f);
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_fWarped);
}


void diffWarp2D(
        const float* f,
        const float* u,
        const float* v,
        float* diffx,
        float* diffy,
        int shape0,
        int shape1
    ){

    size_t size = shape0 * shape1 * sizeof(float);

    // allocate vectors in device memory
    float *d_f, *d_u, *d_v, *d_diffx, *d_diffy;
    gpuErrchk(hipMalloc(&d_f, size));
    gpuErrchk(hipMalloc(&d_u, size));
    gpuErrchk(hipMalloc(&d_v, size));
    gpuErrchk(hipMalloc(&d_diffx, size));
    gpuErrchk(hipMalloc(&d_diffy, size));

    // copy vectors from host memory to device memory
    gpuErrchk(hipMemcpy(d_f, f, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_u, u, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_v, v, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_diffx, diffx, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_diffy, diffy, size, hipMemcpyHostToDevice));

    // kernel invocation with 16*16 threads per block, and enough blocks
    // to cover the entire length of the vectors
    dim3 threadsPerBlock(16,16);
    dim3 numBlocks((shape0 + 15)/16, (shape1 + 15)/16);
    float coeffsx[] = {
        #include "cubic_2D_coefficients_dx.inc"
    };
    float coeffsy[] = {
        #include "cubic_2D_coefficients_dy.inc"
    };
    float *d_coeffsx;
    float *d_coeffsy;
    gpuErrchk(hipMalloc(&d_coeffsx, 16*16*sizeof(float)));
    gpuErrchk(hipMalloc(&d_coeffsy, 16*16*sizeof(float)));
    gpuErrchk(hipMemcpy(d_coeffsx, coeffsx, 16*16*sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_coeffsy, coeffsy, 16*16*sizeof(float), hipMemcpyHostToDevice));
    cubicWarp2DKernel<<<numBlocks, threadsPerBlock>>>(
        d_f,
        d_u,
        d_v,
        d_diffx,
        shape0,
        shape1,
        d_coeffsx
    );
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    cubicWarp2DKernel<<<numBlocks, threadsPerBlock>>>(
        d_f,
        d_u,
        d_v,
        d_diffy,
        shape0,
        shape1,
        d_coeffsy
    );
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    hipFree(d_coeffsx);
    hipFree(d_coeffsy);

    // copy the result back to the host
    gpuErrchk(hipMemcpy(diffx, d_diffx, size, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(diffy, d_diffy, size, hipMemcpyDeviceToHost));

    // release the device memory
    hipFree(d_f);
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_diffx);
    hipFree(d_diffy);
}


void jvpWarp2D(
        const float* f,
        const float* u,
        const float* v,
        const float* input,
        float* output,
        int degree,
        int shape0,
        int shape1
    ){

    size_t size = shape0 * shape1 * sizeof(float);

    // allocate vectors in device memory
    float *d_f, *d_u, *d_v, *d_input, *d_output;
    gpuErrchk(hipMalloc(&d_f, size));
    gpuErrchk(hipMalloc(&d_u, size));
    gpuErrchk(hipMalloc(&d_v, size));
    gpuErrchk(hipMalloc(&d_input, size));
    gpuErrchk(hipMalloc(&d_output, 2*size));

    // copy vectors from host memory to device memory
    gpuErrchk(hipMemcpy(d_f, f, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_u, u, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_v, v, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_input, input, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_output, output, 2*size, hipMemcpyHostToDevice));

    // kernel invocation with 16*16 threads per block, and enough blocks
    // to cover the entire length of the vectors
    dim3 threadsPerBlock(16,16);
    dim3 numBlocks((shape0 + 15)/16, (shape1 + 15)/16);
    if(degree==1){
        jvpxLinearWarp2DKernel<<<numBlocks, threadsPerBlock>>>(
            d_f,
            d_u,
            d_v,
            d_input,
            d_output,
            shape0,
            shape1
        );
        jvpyLinearWarp2DKernel<<<numBlocks, threadsPerBlock>>>(
            d_f,
            d_u,
            d_v,
            d_input,
            d_output + shape0*shape1,
            shape0,
            shape1
        );
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }else if(degree==3){
        float coeffs_dx[] = {
            #include "cubic_2D_coefficients_dx.inc"
        };
        float coeffs_dy[] = {
            #include "cubic_2D_coefficients_dy.inc"
        };
        float *d_coeffs_dx;
        float *d_coeffs_dy;
        gpuErrchk(hipMalloc(&d_coeffs_dx, 16*16*sizeof(float)));
        gpuErrchk(hipMalloc(&d_coeffs_dy, 16*16*sizeof(float)));
        gpuErrchk(hipMemcpy(d_coeffs_dx, coeffs_dx, 16*16*sizeof(float), hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(d_coeffs_dy, coeffs_dy, 16*16*sizeof(float), hipMemcpyHostToDevice));
        jvpCubicWarp2DKernel<<<numBlocks, threadsPerBlock>>>(d_f,
            d_u,
            d_v,
            d_input,
            d_output,
            shape0,
            shape1,
            d_coeffs_dx
        );
        jvpCubicWarp2DKernel<<<numBlocks, threadsPerBlock>>>(d_f,
            d_u,
            d_v,
            d_input,
            d_output + shape0*shape1,
            shape0,
            shape1,
            d_coeffs_dy
        );
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        hipFree(d_coeffs_dx);
        hipFree(d_coeffs_dy);
    }else{
        throw "Only degree 1 and 3 are implemented";
    }

    // copy the result back to the host
    gpuErrchk(hipMemcpy(output, d_output, 2*size, hipMemcpyDeviceToHost));

    // release the device memory
    hipFree(d_f);
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_input);
    hipFree(d_output);
}


void warp3D(
        const float* f,
        const float* u,
        const float* v,
        const float* w,
        float* fWarped,
        int degree,
        int shape0,
        int shape1,
        int shape2
    ){

    size_t size = shape0 * shape1 * shape2 * sizeof(float);

    // allocate vectors in device memory
    float *d_f, *d_u, *d_v, *d_w, *d_fWarped;
    gpuErrchk(hipMalloc(&d_f, size));
    gpuErrchk(hipMalloc(&d_u, size));
    gpuErrchk(hipMalloc(&d_v, size));
    gpuErrchk(hipMalloc(&d_w, size));
    gpuErrchk(hipMalloc(&d_fWarped, size));

    // copy vectors from host memory to device memory
    gpuErrchk(hipMemcpy(d_f, f, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_u, u, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_v, v, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_w, w, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_fWarped, fWarped, size, hipMemcpyHostToDevice));

    // kernel invocation with 8*8*8 threads per block, and enough blocks
    // to cover the entire image
    dim3 threadsPerBlock(8,8,8);
    dim3 numBlocks((shape0 + 7)/8, (shape1 + 7)/8, (shape2 + 7)/8);
    if(degree==1){
        linearWarp3DKernel<<<numBlocks, threadsPerBlock>>>(
            d_f,
            d_u,
            d_v,
            d_w,
            d_fWarped,
            shape0,
            shape1,
            shape2
        );
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }else if(degree==3){
        float coeffs[] = {
            #include "cubic_3D_coefficients.inc"
        };
        float *d_coeffs;
        gpuErrchk(hipMalloc(&d_coeffs, 64*64*sizeof(float)));
        gpuErrchk(hipMemcpy(d_coeffs, coeffs, 64*64*sizeof(float), hipMemcpyHostToDevice));
        cubicWarp3DKernel<<<numBlocks, threadsPerBlock>>>(
            d_f,
            d_u,
            d_v,
            d_w,
            d_fWarped,
            shape0,
            shape1,
            shape2,
            d_coeffs
        );
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        hipFree(d_coeffs);
    }else{
        throw "Only degree 1 and 3 are implemented";
    }

    // copy the result back to the host
    gpuErrchk(hipMemcpy(fWarped, d_fWarped, size, hipMemcpyDeviceToHost));

    // release the device memory
    hipFree(d_f);
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_w);
    hipFree(d_fWarped);
}


void adjointWarp3D(
        const float* fWarped,
        const float* u,
        const float* v,
        const float* w,
        float* f,
        int degree,
        int shape0,
        int shape1,
        int shape2
    ){
    /*
    GPU implementation of 3D adjoint backward image warping along the DVF (u,v,w)
    with rectangular multivariate spline interpolation
    */

    size_t size = shape0 * shape1 * shape2 * sizeof(float);

    // allocate vectors in device memory
    float *d_fWarped, *d_u, *d_v, *d_w, *d_f;
    gpuErrchk(hipMalloc(&d_fWarped, size));
    gpuErrchk(hipMalloc(&d_u, size));
    gpuErrchk(hipMalloc(&d_v, size));
    gpuErrchk(hipMalloc(&d_w, size));
    gpuErrchk(hipMalloc(&d_f, size));

    // copy vectors from host memory to device memory
    gpuErrchk(hipMemcpy(d_fWarped, fWarped, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_u, u, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_v, v, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_w, w, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_f, f, size, hipMemcpyHostToDevice));

    // kernel invocation with 8*8*8 threads per block, and enough blocks
    // to cover the entire length of the vectors
    dim3 threadsPerBlock(8,8,8);
    dim3 numBlocks((shape0 + 7)/8, (shape1 + 7)/8, (shape2 + 7)/8);

    if(degree==1){
        adjointLinearWarp3DKernel<<<numBlocks, threadsPerBlock>>>(
            d_fWarped,
            d_u,
            d_v,
            d_w,
            d_f,
            shape0,
            shape1,
            shape2
        );
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }else if(degree==3){
        float coeffs[] = {
            #include "cubic_3D_coefficients.inc"
        };
        float *d_coeffs;
        gpuErrchk(hipMalloc(&d_coeffs, 64*64*sizeof(float)));
        gpuErrchk(hipMemcpy(d_coeffs, coeffs, 64*64*sizeof(float), hipMemcpyHostToDevice));
        adjointCubicWarp3DKernel<<<numBlocks, threadsPerBlock>>>(
            d_fWarped,
            d_u,
            d_v,
            d_w,
            d_f,
            shape0,
            shape1,
            shape2,
            d_coeffs
        );
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        hipFree(d_coeffs);
    }else{
        throw "Only degree 1 and 3 are implemented";
    }
    // copy the result back to the host
    gpuErrchk(hipMemcpy(f, d_f, size, hipMemcpyDeviceToHost));

    // release the device memory
    hipFree(d_f);
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_w);
    hipFree(d_fWarped);
}


void diffWarp3D(
        const float* f,
        const float* u,
        const float* v,
        const float* w,
        float* diffx,
        float* diffy,
        float* diffz,
        int shape0,
        int shape1,
        int shape2
    ){

    size_t size = shape0 * shape1 * shape2 * sizeof(float);

    // allocate vectors in device memory
    float *d_f, *d_u, *d_v, *d_w, *d_diffx, *d_diffy, *d_diffz;
    gpuErrchk(hipMalloc(&d_f, size));
    gpuErrchk(hipMalloc(&d_u, size));
    gpuErrchk(hipMalloc(&d_v, size));
    gpuErrchk(hipMalloc(&d_w, size));
    gpuErrchk(hipMalloc(&d_diffx, size));
    gpuErrchk(hipMalloc(&d_diffy, size));
    gpuErrchk(hipMalloc(&d_diffz, size));

    // copy vectors from host memory to device memory
    gpuErrchk(hipMemcpy(d_f, f, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_u, u, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_v, v, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_w, w, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_diffx, diffx, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_diffy, diffy, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_diffz, diffz, size, hipMemcpyHostToDevice));

    // kernel invocation with 16*16 threads per block, and enough blocks
    // to cover the entire length of the vectors
    dim3 threadsPerBlock(8,8,8);
    dim3 numBlocks((shape0 + 7)/8, (shape1 + 7)/8, (shape2 + 7)/8);
    float coeffsx[] = {
        #include "cubic_3D_coefficients_dx.inc"
    };
    float coeffsy[] = {
        #include "cubic_3D_coefficients_dy.inc"
    };
    float coeffsz[] = {
        #include "cubic_3D_coefficients_dz.inc"
    };
    float *d_coeffsx;
    float *d_coeffsy;
    float *d_coeffsz;
    gpuErrchk(hipMalloc(&d_coeffsx, 64*64*sizeof(float)));
    gpuErrchk(hipMalloc(&d_coeffsy, 64*64*sizeof(float)));
    gpuErrchk(hipMalloc(&d_coeffsz, 64*64*sizeof(float)));
    gpuErrchk(hipMemcpy(d_coeffsx, coeffsx, 64*64*sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_coeffsy, coeffsy, 64*64*sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_coeffsz, coeffsz, 64*64*sizeof(float), hipMemcpyHostToDevice));
    cubicWarp3DKernel<<<numBlocks, threadsPerBlock>>>(
        d_f,
        d_u,
        d_v,
        d_w,
        d_diffx,
        shape0,
        shape1,
        shape2,
        d_coeffsx
    );
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    cubicWarp3DKernel<<<numBlocks, threadsPerBlock>>>(
        d_f,
        d_u,
        d_v,
        d_w,
        d_diffy,
        shape0,
        shape1,
        shape2,
        d_coeffsy
    );
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    cubicWarp3DKernel<<<numBlocks, threadsPerBlock>>>(
        d_f,
        d_u,
        d_v,
        d_w,
        d_diffz,
        shape0,
        shape1,
        shape2,
        d_coeffsz
    );
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    hipFree(d_coeffsx);
    hipFree(d_coeffsy);
    hipFree(d_coeffsz);

    // copy the result back to the host
    gpuErrchk(hipMemcpy(diffx, d_diffx, size, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(diffy, d_diffy, size, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(diffz, d_diffz, size, hipMemcpyDeviceToHost));

    // release the device memory
    hipFree(d_f);
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_w);
    hipFree(d_diffx);
    hipFree(d_diffy);
    hipFree(d_diffz);
}


void partialDiffWarp3D(
        const float* f,
        const float* u,
        const float* v,
        const float* w,
        int to,
        float* diff,
        int shape0,
        int shape1,
        int shape2
    ){

    size_t size = shape0 * shape1 * shape2 * sizeof(float);

    // allocate vectors in device memory
    float *d_f, *d_u, *d_v, *d_w, *d_diff;
    gpuErrchk(hipMalloc(&d_f, size));
    gpuErrchk(hipMalloc(&d_u, size));
    gpuErrchk(hipMalloc(&d_v, size));
    gpuErrchk(hipMalloc(&d_w, size));
    gpuErrchk(hipMalloc(&d_diff, size));

    // copy vectors from host memory to device memory
    gpuErrchk(hipMemcpy(d_f, f, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_u, u, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_v, v, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_w, w, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_diff, diff, size, hipMemcpyHostToDevice));

    // kernel invocation with 8*8*8 threads per block, and enough blocks
    // to cover the entire length of the vectors
    dim3 threadsPerBlock(8,8,8);
    dim3 numBlocks((shape0 + 7)/8, (shape1 + 7)/8, (shape2 + 7)/8);
    float *coeffs;
    if(to == 0){
        float coeffsx[] = {
            #include "cubic_3D_coefficients_dx.inc"
        };
        coeffs = coeffsx;
    }else if(to == 1){
        float coeffsy[] = {
            #include "cubic_3D_coefficients_dy.inc"
        };
        coeffs = coeffsy;
    }else{
        float coeffsz[] = {
            #include "cubic_3D_coefficients_dz.inc"
        };
        coeffs = coeffsz;
    }
    float *d_coeffs;
    gpuErrchk(hipMalloc(&d_coeffs, 64*64*sizeof(float)));
    gpuErrchk(hipMemcpy(d_coeffs, coeffs, 64*64*sizeof(float), hipMemcpyHostToDevice));
    cubicWarp3DKernel<<<numBlocks, threadsPerBlock>>>(
        d_f,
        d_u,
        d_v,
        d_w,
        d_diff,
        shape0,
        shape1,
        shape2,
        d_coeffs
    );
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    hipFree(d_coeffs);

    // copy the result back to the host
    gpuErrchk(hipMemcpy(diff, d_diff, size, hipMemcpyDeviceToHost));

    // release the device memory
    hipFree(d_f);
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_w);
    hipFree(d_diff);
}


void jvpWarp3D(
        const float* f,
        const float* u,
        const float* v,
        const float* w,
        const float* input,
        float* output,
        int shape0,
        int shape1,
        int shape2
    ){

    size_t size = shape0 * shape1 * shape2 * sizeof(float);

    // allocate vectors in device memory
    float *d_f, *d_u, *d_v, *d_w, *d_input, *d_output;
    gpuErrchk(hipMalloc(&d_f, size));
    gpuErrchk(hipMalloc(&d_u, size));
    gpuErrchk(hipMalloc(&d_v, size));
    gpuErrchk(hipMalloc(&d_w, size));
    gpuErrchk(hipMalloc(&d_input, size));
    gpuErrchk(hipMalloc(&d_output, 3*size));

    // copy vectors from host memory to device memory
    gpuErrchk(hipMemcpy(d_f, f, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_u, u, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_v, v, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_w, w, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_input, input, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_output, output, 3*size, hipMemcpyHostToDevice));

    // kernel invocation with 8*8*8 threads per block, and enough blocks
    // to cover the entire length of the vectors
    dim3 threadsPerBlock(8,8,8);
    dim3 numBlocks((shape0 + 7)/8, (shape1 + 7)/8, (shape2 + 7)/8);

    float coeffs_dx[] = {
        #include "cubic_3D_coefficients_dx.inc"
    };
    float coeffs_dy[] = {
        #include "cubic_3D_coefficients_dy.inc"
    };
    float coeffs_dz[] = {
        #include "cubic_3D_coefficients_dz.inc"
    };
    float *d_coeffs_dx;
    float *d_coeffs_dy;
    float *d_coeffs_dz;
    gpuErrchk(hipMalloc(&d_coeffs_dx, 64*64*sizeof(float)));
    gpuErrchk(hipMalloc(&d_coeffs_dy, 64*64*sizeof(float)));
    gpuErrchk(hipMalloc(&d_coeffs_dz, 64*64*sizeof(float)));
    gpuErrchk(hipMemcpy(d_coeffs_dx, coeffs_dx, 64*64*sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_coeffs_dy, coeffs_dy, 64*64*sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_coeffs_dz, coeffs_dz, 64*64*sizeof(float), hipMemcpyHostToDevice));
    jvpCubicWarp3DKernel<<<numBlocks, threadsPerBlock>>>(
        d_f,
        d_u,
        d_v,
        d_w,
        d_input,
        d_output,
        shape0,
        shape1,
        shape2,
        d_coeffs_dx
    );
    jvpCubicWarp3DKernel<<<numBlocks, threadsPerBlock>>>(
        d_f,
        d_u,
        d_v,
        d_w,
        d_input,
        d_output + shape0*shape1*shape2,
        shape0,
        shape1,
        shape2,
        d_coeffs_dy
    );
    jvpCubicWarp3DKernel<<<numBlocks, threadsPerBlock>>>(
        d_f,
        d_u,
        d_v,
        d_w,
        d_input,
        d_output + shape0*shape1*shape2*2,
        shape0,
        shape1,
        shape2,
        d_coeffs_dz
    );
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    hipFree(d_coeffs_dx);
    hipFree(d_coeffs_dy);
    hipFree(d_coeffs_dz);

    // copy the result back to the host
    gpuErrchk(hipMemcpy(output, d_output, 3*size, hipMemcpyDeviceToHost));

    // release the device memory
    hipFree(d_f);
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_w);
    hipFree(d_input);
    hipFree(d_output);
}


void jvpWarp3DY(
        const float* f,
        const float* u,
        const float* v,
        const float* w,
        const float* input,
        float* output,
        int shape0,
        int shape1,
        int shape2
    ){

    size_t size = shape0 * shape1 * shape2 * sizeof(float);

    // allocate vectors in device memory
    float *d_f, *d_u, *d_v, *d_w, *d_input, *d_output;
    gpuErrchk(hipMalloc(&d_f, size));
    gpuErrchk(hipMalloc(&d_u, size));
    gpuErrchk(hipMalloc(&d_v, size));
    gpuErrchk(hipMalloc(&d_w, size));
    gpuErrchk(hipMalloc(&d_input, size));
    gpuErrchk(hipMalloc(&d_output, size));

    // copy vectors from host memory to device memory
    gpuErrchk(hipMemcpy(d_f, f, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_u, u, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_v, v, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_w, w, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_input, input, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_output, output, size, hipMemcpyHostToDevice));

    // kernel invocation with 8*8*8 threads per block, and enough blocks
    // to cover the entire length of the vectors
    dim3 threadsPerBlock(8,8,8);
    dim3 numBlocks((shape0 + 7)/8, (shape1 + 7)/8, (shape2 + 7)/8);

    float coeffs_dy[] = {
        #include "cubic_3D_coefficients_dy.inc"
    };
    float *d_coeffs_dy;
    gpuErrchk(hipMalloc(&d_coeffs_dy, 64*64*sizeof(float)));
    gpuErrchk(hipMemcpy(d_coeffs_dy, coeffs_dy, 64*64*sizeof(float), hipMemcpyHostToDevice));
    jvpCubicWarp3DKernel<<<numBlocks, threadsPerBlock>>>(
        d_f,
        d_u,
        d_v,
        d_w,
        d_input,
        d_output,
        shape0,
        shape1,
        shape2,
        d_coeffs_dy
    );

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    hipFree(d_coeffs_dy);

    // copy the result back to the host
    gpuErrchk(hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost));

    // release the device memory
    hipFree(d_f);
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_w);
    hipFree(d_input);
    hipFree(d_output);
}