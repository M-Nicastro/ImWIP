/*
:file:      utils.cu
:brief:     utils
:date:      1 AUG 2022
:author:    Jens Renders
            imec-Visionlab
            University of Antwerp
            jens.renders@uantwerpen.be
*/
#include <stdio.h>
#include <utils.hu>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


int getDevice(){
    int* device;
    gpuErrchk(hipGetDevice(device));
    return *device;
}


void setDevice(int device){
    gpuErrchk(hipSetDevice(device));
}


int getDeviceCount(){
    int* count;
    gpuErrchk(hipGetDeviceCount(count));
    return *count;
}

std::string getDeviceName(int device){
    hipDeviceProp_t* prop;
    gpuErrchk(hipGetDeviceProperties(prop, device));
    std::string name(prop->name);
    return name;
}