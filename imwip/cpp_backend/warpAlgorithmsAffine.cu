/*
:file:      warpAlgorithmsAffine.cu
:brief:     Affine warping algorithms
:date:      20 DEC 2021
:author:    Jens Renders
            imec-Visionlab
            University of Antwerp
            jens.renders@uantwerpen.be
*/
#include <stdio.h>

#include <warpKernelsAffine.cu>
#include <warpAlgorithmsAffine.hu>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void affineWarp2D(
        const float* f,
        const float* A,
        const float* b,
        float* fWarped,
        int degree,
        int shape0,
        int shape1
    ){

    size_t size = shape0 * shape1 * sizeof(float);

    // allocate vectors in device memory
    float *d_f, *d_A, *d_b, *d_fWarped;
    gpuErrchk(hipMalloc(&d_f, size));
    gpuErrchk(hipMalloc(&d_A, 4 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_b, 2 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_fWarped, size));

    // copy vectors from host memory to device memory
    gpuErrchk(hipMemcpy(d_f, f, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_A, A, 4 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_b, b, 2 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_fWarped, fWarped, size, hipMemcpyHostToDevice));

    // kernel invocation with 16*16 threads per block, and enough blocks
    // to cover the entire length of the vectors
    dim3 threadsPerBlock(16,16);
    dim3 numBlocks((shape0 + 15)/16, (shape1 + 15)/16);
    if(degree==1){
        affineLinearWarp2DKernel<<<numBlocks, threadsPerBlock>>>(
            d_f,
            d_A,
            d_b,
            d_fWarped,
            shape0,
            shape1
        );
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }else if(degree==3){
        float coeffs[] = {
            #include "cubic_2D_coefficients.inc"
        };
        float *d_coeffs;
        gpuErrchk(hipMalloc(&d_coeffs, 16*16*sizeof(float)));
        gpuErrchk(hipMemcpy(d_coeffs, coeffs, 16*16*sizeof(float), hipMemcpyHostToDevice));
        affineCubicWarp2DKernel<<<numBlocks, threadsPerBlock>>>(
            d_f,
            d_A,
            d_b,
            d_fWarped,
            shape0,
            shape1,
            d_coeffs
        );
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        hipFree(d_coeffs);
    }else{
        throw "Only degree 1 and 3 are implemented";
    }

    // copy the result back to the host
    gpuErrchk(hipMemcpy(fWarped, d_fWarped, size, hipMemcpyDeviceToHost));

    // release the device memory
    hipFree(d_f);
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_fWarped);
}


void adjointAffineWarp2D(
        const float* fWarped,
        const float* A,
        const float* b,
        float* f,
        int degree,
        int shape0,
        int shape1
    ){

    /*
    GPU implementation of 2D adjoint backward image warping along the DVF (u,v)
    with rectangular multivariate spline interpolation
    */


    size_t size = shape0 * shape1 * sizeof(float);

    // allocate vectors in device memory
    float *d_fWarped, *d_A, *d_b, *d_f;
    gpuErrchk(hipMalloc(&d_fWarped, size));
    gpuErrchk(hipMalloc(&d_A, 4 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_b, 2 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_f, size));

    // copy vectors from host memory to device memory
    gpuErrchk(hipMemcpy(d_fWarped, fWarped, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_A, A, 4 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_b, b, 2 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_f, f, size, hipMemcpyHostToDevice));

    // kernel invocation with 16*16 threads per block, and enough blocks
    // to cover the entire length of the vectors
    dim3 threadsPerBlock(16,16);
    dim3 numBlocks((shape0 + 15)/16, (shape1 + 15)/16);
    if(degree==1){
        adjointAffineLinearWarp2DKernel<<<numBlocks, threadsPerBlock>>>(
            d_fWarped,
            d_A,
            d_b,
            d_f,
            shape0,
            shape1
        );
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }else if(degree==3){
        float coeffs[] = {
            #include "cubic_2D_coefficients.inc"
        };
        float *d_coeffs;
        gpuErrchk(hipMalloc(&d_coeffs, 16*16*sizeof(float)));
        gpuErrchk(hipMemcpy(d_coeffs, coeffs, 16*16*sizeof(float), hipMemcpyHostToDevice));
        adjointAffineCubicWarp2DKernel<<<numBlocks, threadsPerBlock>>>(
            d_fWarped,
            d_A,
            d_b,
            d_f,
            shape0,
            shape1,
            d_coeffs
        );
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        hipFree(d_coeffs);
    }else{
        throw "Only degree 1 and 3 are implemented";
    }

    // copy the result back to the host
    gpuErrchk(hipMemcpy(f, d_f, size, hipMemcpyDeviceToHost));

    // release the device memory
    hipFree(d_f);
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_fWarped);
}


void gradAffineWarp2D(
        const float* f,
        const float* A,
        const float* b,
        float* gradx,
        float* grady,
        int shape0,
        int shape1
    ){

    size_t size = shape0 * shape1 * sizeof(float);

    // allocate vectors in device memory
    float *d_f, *d_A, *d_b, *d_gradx, *d_grady;
    gpuErrchk(hipMalloc(&d_f, size));
    gpuErrchk(hipMalloc(&d_A, 4 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_b, 2 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_gradx, size));
    gpuErrchk(hipMalloc(&d_grady, size));

    // copy vectors from host memory to device memory
    gpuErrchk(hipMemcpy(d_f, f, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_A, A, 4 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_b, b, 2 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_gradx, gradx, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_grady, grady, size, hipMemcpyHostToDevice));

    // kernel invocation with 16*16 threads per block, and enough blocks
    // to cover the entire length of the vectors
    dim3 threadsPerBlock(16,16);
    dim3 numBlocks((shape0 + 15)/16, (shape1 + 15)/16);
    float coeffsx[] = {
        #include "cubic_2D_coefficients_dx.inc"
    };
    float coeffsy[] = {
        #include "cubic_2D_coefficients_dy.inc"
    };
    float *d_coeffsx;
    float *d_coeffsy;
    gpuErrchk(hipMalloc(&d_coeffsx, 16*16*sizeof(float)));
    gpuErrchk(hipMalloc(&d_coeffsy, 16*16*sizeof(float)));
    gpuErrchk(hipMemcpy(d_coeffsx, coeffsx, 16*16*sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_coeffsy, coeffsy, 16*16*sizeof(float), hipMemcpyHostToDevice));
    affineCubicWarp2DKernel<<<numBlocks, threadsPerBlock>>>(
        d_f,
        d_A,
        d_b,
        d_gradx,
        shape0,
        shape1,
        d_coeffsx
    );
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    affineCubicWarp2DKernel<<<numBlocks, threadsPerBlock>>>(
        d_f,
        d_A,
        d_b,
        d_grady,
        shape0,
        shape1,
        d_coeffsy
    );
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    hipFree(d_coeffsx);
    hipFree(d_coeffsy);

    // copy the result back to the host
    gpuErrchk(hipMemcpy(gradx, d_gradx, size, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(grady, d_grady, size, hipMemcpyDeviceToHost));

    // release the device memory
    hipFree(d_f);
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_gradx);
    hipFree(d_grady);
}


void affineWarp3D(
        const float* f,
        const float* A,
        const float* b,
        float* fWarped,
        int degree,
        int shape0,
        int shape1,
        int shape2
    ){

    size_t size = shape0 * shape1 * shape2 * sizeof(float);

    // allocate vectors in device memory
    float *d_f, *d_A, *d_b, *d_fWarped;
    gpuErrchk(hipMalloc(&d_f, size));
    gpuErrchk(hipMalloc(&d_A, 9 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_b, 3 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_fWarped, size));

    // copy vectors from host memory to device memory
    gpuErrchk(hipMemcpy(d_f, f, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_A, A, 9 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_b, b, 3 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_fWarped, fWarped, size, hipMemcpyHostToDevice));

    // kernel invocation with 8*8*8 threads per block, and enough blocks
    // to cover the entire image
    dim3 threadsPerBlock(8,8,8);
    dim3 numBlocks((shape2 + 7)/8, (shape1 + 7)/8, (shape0 + 7)/8); //faster order
    if(degree==1){
        affineLinearWarp3DKernel<<<numBlocks, threadsPerBlock>>>(
            d_f,
            d_A,
            d_b,
            d_fWarped,
            shape0,
            shape1,
            shape2
        );
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }else if(degree==3){
        float coeffs[] = {
            #include "cubic_3D_coefficients.inc"
        };
        float *d_coeffs;
        gpuErrchk(hipMalloc(&d_coeffs, 64*64*sizeof(float)));
        gpuErrchk(hipMemcpy(d_coeffs, coeffs, 64*64*sizeof(float), hipMemcpyHostToDevice));
        affineCubicWarp3DKernel<<<numBlocks, threadsPerBlock>>>(
            d_f,
            d_A,
            d_b,
            d_fWarped,
            shape0,
            shape1,
            shape2,
            d_coeffs
        );
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        hipFree(d_coeffs);
    }else{
        throw "Only degree 1 and 3 are implemented";
    }

    // copy the result back to the host
    gpuErrchk(hipMemcpy(fWarped, d_fWarped, size, hipMemcpyDeviceToHost));

    // release the device memory
    hipFree(d_f);
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_fWarped);
}


void adjointAffineWarp3D(
        const float* fWarped,
        const float* A,
        const float* b,
        float* f,
        int degree,
        int shape0,
        int shape1,
        int shape2
){
    /*
    GPU implementation of 3D adjoint backward image warping along the DVF (u,v,w)
    with rectangular multivariate spline interpolation
    */

    size_t size = shape0 * shape1 * shape2 * sizeof(float);

    // allocate vectors in device memory
    float *d_fWarped, *d_A, *d_b, *d_f;
    gpuErrchk(hipMalloc(&d_fWarped, size));
    gpuErrchk(hipMalloc(&d_A, 9 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_b, 3 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_f, size));

    // copy vectors from host memory to device memory
    gpuErrchk(hipMemcpy(d_fWarped, fWarped, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_A, A, 9 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_b, b, 3 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_f, f, size, hipMemcpyHostToDevice));

    // kernel invocation with 8*8*8 threads per block, and enough blocks
    // to cover the entire length of the vectors
    dim3 threadsPerBlock(8,8,8);
    dim3 numBlocks((shape2 + 7)/8, (shape1 + 7)/8, (shape0 + 7)/8); //faster order

    if(degree==1){
        adjointAffineLinearWarp3DKernel<<<numBlocks, threadsPerBlock>>>(
            d_fWarped,
            d_A,
            d_b,
            d_f,
            shape0,
            shape1,
            shape2
        );
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }else if(degree==3){
        float coeffs[] = {
            #include "cubic_3D_coefficients.inc"
        };
        float *d_coeffs;
        gpuErrchk(hipMalloc(&d_coeffs, 64*64*sizeof(float)));
        gpuErrchk(hipMemcpy(d_coeffs, coeffs, 64*64*sizeof(float), hipMemcpyHostToDevice));
        adjointAffineCubicWarp3DKernel<<<numBlocks, threadsPerBlock>>>(
            d_fWarped,
            d_A,
            d_b,
            d_f,
            shape0,
            shape1,
            shape2,
            d_coeffs
        );
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        hipFree(d_coeffs);
    }else{
        throw "Only degree 1 and 3 are implemented";
    }
    // copy the result back to the host
    gpuErrchk(hipMemcpy(f, d_f, size, hipMemcpyDeviceToHost));

    // release the device memory
    hipFree(d_f);
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_fWarped);
}


void gradAffineWarp3D(
        const float* f,
        const float* A,
        const float* b,
        float* gradx,
        float* grady,
        float* gradz,
        int shape0,
        int shape1,
        int shape2
    ){

    size_t size = shape0 * shape1 * shape2 * sizeof(float);

    // allocate vectors in device memory
    float *d_f, *d_A, *d_b, *d_gradx, *d_grady, *d_gradz;
    gpuErrchk(hipMalloc(&d_f, size));
    gpuErrchk(hipMalloc(&d_A, 9 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_b, 3 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_gradx, size));
    gpuErrchk(hipMalloc(&d_grady, size));
    gpuErrchk(hipMalloc(&d_gradz, size));

    // copy vectors from host memory to device memory
    gpuErrchk(hipMemcpy(d_f, f, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_A, A, 9 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_b, b, 3 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_gradx, gradx, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_grady, grady, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_gradz, gradz, size, hipMemcpyHostToDevice));

    // kernel invocation with 16*16 threads per block, and enough blocks
    // to cover the entire length of the vectors
    dim3 threadsPerBlock(8,8,8);
    dim3 numBlocks((shape2 + 7)/8, (shape1 + 7)/8, (shape0 + 7)/8); //faster order
    float coeffsx[] = {
        #include "cubic_3D_coefficients_dx.inc"
    };
    float coeffsy[] = {
        #include "cubic_3D_coefficients_dy.inc"
    };
    float coeffsz[] = {
        #include "cubic_3D_coefficients_dz.inc"
    };
    float *d_coeffsx;
    float *d_coeffsy;
    float *d_coeffsz;
    gpuErrchk(hipMalloc(&d_coeffsx, 64*64*sizeof(float)));
    gpuErrchk(hipMalloc(&d_coeffsy, 64*64*sizeof(float)));
    gpuErrchk(hipMalloc(&d_coeffsz, 64*64*sizeof(float)));
    gpuErrchk(hipMemcpy(d_coeffsx, coeffsx, 64*64*sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_coeffsy, coeffsy, 64*64*sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_coeffsz, coeffsz, 64*64*sizeof(float), hipMemcpyHostToDevice));
    affineCubicWarp3DKernel<<<numBlocks, threadsPerBlock>>>(
        d_f,
        d_A,
        d_b,
        d_gradx,
        shape0,
        shape1,
        shape2,
        d_coeffsx
    );
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    affineCubicWarp3DKernel<<<numBlocks, threadsPerBlock>>>(
        d_f,
        d_A,
        d_b,
        d_grady,
        shape0,
        shape1,
        shape2,
        d_coeffsy
    );
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    affineCubicWarp3DKernel<<<numBlocks, threadsPerBlock>>>(
        d_f,
        d_A,
        d_b,
        d_gradz,
        shape0,
        shape1,
        shape2,
        d_coeffsz
    );
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    hipFree(d_coeffsx);
    hipFree(d_coeffsy);
    hipFree(d_coeffsz);

    // copy the result back to the host
    gpuErrchk(hipMemcpy(gradx, d_gradx, size, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(grady, d_grady, size, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(gradz, d_gradz, size, hipMemcpyDeviceToHost));

    // release the device memory
    hipFree(d_f);
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_gradx);
    hipFree(d_grady);
    hipFree(d_gradz);
}